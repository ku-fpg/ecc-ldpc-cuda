// NOTE: h_ prefix means "host"   (CPU)
//       d_ prefix means "device" (GPU)
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
using namespace std;

// GPU code
__global__ void soft_to_hard(double* soft, int* hard) {
  int ix = (blockDim.x * blockIdx.x) + threadIdx.x;

  hard[ix] = (soft[ix] > 0);  // XXX: Is this faster than an 'if'?
}


// CPU code
int main(int argc, char* argv[]) {
  if (argc != 2) {
    cout << "Wrong number of arguments. Expected 1, got " << (argc-1)
         << endl;
    return 1;
  }

  // Read values from input file
  ifstream file(argv[1]);
  int softCount;
  file >> softCount;

  double* h_soft = new double[softCount];

  for (int i = 0; i < softCount; ++i) {
    file >> h_soft[i];
  }

  file.close();

  // Copy values to device memory
  double* d_soft;
  hipMalloc((void**)&d_soft, softCount*sizeof(double));
  hipMemcpy(d_soft, h_soft, softCount*sizeof(double), hipMemcpyHostToDevice);

  int* d_hard;
  hipMalloc((void**)&d_hard, softCount*sizeof(int));

  // Set up the computational grid
  int threadsPerBlock = 2;
  int blocksPerGrid   = (softCount + threadsPerBlock - 1) / threadsPerBlock;

  // Launch kernel
  soft_to_hard<<<blocksPerGrid, threadsPerBlock>>>(d_soft, d_hard);

  // Copy results to host memory
  int* h_hard = new int[softCount];
  hipMemcpy(h_hard, d_hard, softCount*sizeof(int), hipMemcpyDeviceToHost);

  // Print results
  for (int i = 0; i < softCount; ++i) {
    cout << h_hard[i] << ' ';
  }
  cout << endl;

  hipFree(d_soft);
  hipFree(d_hard);
  delete[] h_soft;
  delete[] h_hard;

  return 0;
}

